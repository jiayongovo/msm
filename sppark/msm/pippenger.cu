#include "hip/hip_runtime.h"
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>


#ifndef WARP_SZ
# define WARP_SZ 32
#endif

#ifndef NTHREADS
# define NTHREADS 64
#endif
#if NTHREADS < 32 || (NTHREADS & (NTHREADS-1)) != 0
# error "bad NTHREADS value"
#endif

constexpr static int log2(int n)
{   int ret=0; while (n>>=1) ret++; return ret;   }

static const int NTHRBITS = log2(NTHREADS);

#ifndef NBITS
# define NBITS 253
#endif
#ifndef WBITS
# define WBITS 16
#endif
#define NWINS 16  // ((NBITS+WBITS-1)/WBITS)   // ceil(NBITS/WBITS)

#ifndef LARGE_L1_CODE_CACHE
# define LARGE_L1_CODE_CACHE 0
#endif

__global__
void pre_compute(affine_t* pre_points, size_t npoints);

__global__
void jy_pre_compute(affine_t* pre_points, size_t npoints);


__global__
void process_scalar_1(uint16_t* scalar, uint32_t* scalar_tuple,
                      uint32_t* d_scalar_map, uint32_t* point_idx, size_t npoints);
__global__
void jy_process_scalar_1(uint16_t* scalar, uint32_t* scalar_tuple,
                         uint32_t* point_idx, size_t npoints);

__global__
void process_scalar_2(uint32_t* scalar_tuple_out,
                      uint16_t* bucket_idx, size_t npoints);

__global__
void bucket_inf(bucket_t *buckets);

// v1.1
__global__
void bucket_acc(uint32_t* scalar_tuple_out, uint16_t* bucket_idx, uint32_t* point_idx_out,
                affine_t* pre_points, bucket_t *buckets_pre,
                uint16_t* bucket_idx_pre_vector, uint16_t* bucket_idx_pre_used,
                uint32_t* bucket_idx_pre_offset, size_t npoints);

__global__
void bucket_acc_2(bucket_t *buckets_pre, uint16_t* bucket_idx_pre_vector, uint16_t* bucket_idx_pre_used,
                  uint32_t* bucket_idx_pre_offset, bucket_t *buckets, uint32_t upper_tnum, size_t npoints);

__global__
void bucket_agg_1(bucket_t *buckets);

__global__
void bucket_agg_2(bucket_t *buckets);

__global__
void recursive_sum(bucket_t *buckets, bucket_t *res);


#ifdef __CUDA_ARCH__

#include <hip/hip_cooperative_groups.h>

static __shared__ bucket_t bucket_acc_smem[NTHREADS * 2];

// Transposed scalar_t
class scalar_T {
    uint32_t val[sizeof(scalar_t)/sizeof(uint32_t)][WARP_SZ];

public:
    __device__ uint32_t& operator[](size_t i)              { return val[i][0]; }
    __device__ const uint32_t& operator[](size_t i) const  { return val[i][0]; }
    __device__ scalar_T& operator=(const scalar_t& rhs)
    {
        for (size_t i = 0; i < sizeof(scalar_t)/sizeof(uint32_t); i++)
            val[i][0] = rhs[i];
        return *this;
    }
};

class scalars_T {
    scalar_T* ptr;

public:
    __device__ scalars_T(void* rhs) { ptr = (scalar_T*)rhs; }
    __device__ scalar_T& operator[](size_t i)
    {   return *(scalar_T*)&(&ptr[i/WARP_SZ][0])[i%WARP_SZ];   }
    __device__ const scalar_T& operator[](size_t i) const
    {   return *(const scalar_T*)&(&ptr[i/WARP_SZ][0])[i%WARP_SZ];   }
};

constexpr static __device__ int dlog2(int n)
{   int ret=0; while (n>>=1) ret++; return ret;   }


#if WBITS==16
template<class scalar_t>
static __device__ int get_wval(const scalar_t& d, uint32_t off, uint32_t bits)
{
    uint32_t ret = d[off/32];
    return (ret >> (off%32)) & ((1<<bits) - 1);
}
#else
template<class scalar_t>
static __device__ int get_wval(const scalar_t& d, uint32_t off, uint32_t bits)
{
    uint32_t top = off + bits - 1;
    uint64_t ret = ((uint64_t)d[top/32] << 32) | d[off/32];

    return (int)(ret >> (off%32)) & ((1<<bits) - 1);
}
#endif


static __device__ uint32_t max_bits(uint32_t scalar)
{
    uint32_t max = 32;
    return max;
}

static __device__ bool test_bit(uint32_t scalar, uint32_t bitno)
{
    if (bitno >= 32)
        return false;
    return ((scalar >> bitno) & 0x1);
}

template<class bucket_t>
static __device__ void mul(bucket_t& res, const bucket_t& base, uint32_t scalar)
{
    res.inf();

    bool found_one = false;
    uint32_t mb = max_bits(scalar);
    for (int32_t i = mb - 1; i >= 0; --i)
    {
        if (found_one)
        {
            res.add(res);
        }

        if (test_bit(scalar, i))
        {
            found_one = true;
            res.add(base);
        }
    }
}

__global__
void pre_compute(affine_t* pre_points, size_t npoints) {
    // NWINS * config.N  NTHREADS
    // blockDim.x 表示每个线程块中的线程数量
    // gridDim.x 表示网格中线程块的数量
    // blockIdx.x 表示当前线程块的索引，blockDim.x 表示每个线程块中的线程数量，threadIdx.x 表示当前线程在其线程块中的索引。
    const uint32_t tnum = blockDim.x * gridDim.x;
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    bucket_t Pi_xyzz;
    for (uint32_t i = tid; i < npoints; i += tnum) {
        affine_t* Pi = pre_points + i;
        Pi_xyzz = *Pi;
        for (int j = 1; j < 7; j++) {
            Pi = Pi + npoints;
            Pi_xyzz.dbl();

            Pi_xyzz.xyzz_to_affine(*Pi);
        }
    }
}

// jy_msm 点预计算
__global__
void jy_pre_compute(affine_t* pre_points, size_t npoints) {
    const uint32_t tnum = blockDim.x * gridDim.x;
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t num = (NWINS % 2 ==0 ? NWINS - 2 : NWINS - 1) / 2 ;

    bucket_t Pi_xyzz;
    for (uint32_t i = tid; i < npoints; i += tnum) {
        affine_t* Pi = pre_points + i;
        Pi_xyzz = *Pi;
        for (int j = 1; j <= num; j++) {
            uint32_t pow = 2 * j * WBITS;
            Pi = Pi + npoints;
             for(uint32_t k=0;k<pow;k++)
                Pi_xyzz.dbl();
            Pi_xyzz.xyzz_to_affine(*Pi);
        }
    }
}

// 把输进来的scalar看作是u16
__global__
void process_scalar_1(uint16_t* scalar, uint32_t* scalar_tuple,
                      uint32_t* d_scalar_map, uint32_t* point_idx, size_t npoints) {

    const uint32_t tnum = blockDim.x * gridDim.x;
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // 每个线程分配到一个标量的划分
    // 第 i 个标量
    for (int i = tid; i < npoints; i += tnum) {
        // 因为把他看作是u16 因此偏移需要加 2^16 * i
        // 当前线程处理的起始标量 ki0
        uint16_t* cur_scalar_ptr = scalar + (i << 4);
        // 获得标量值
        uint32_t cur_scalar = (uint32_t)(*cur_scalar_ptr);  // uint32_t instead of uint16_t, specifically for 0x10000
        // 根据 ki0 查找获得浮点形式 kij
        //
        scalar_tuple[i] = d_scalar_map[cur_scalar];

        point_idx[i] = i;
        // j 放进去
        for (int j = i + npoints; j < NWINS * npoints; j += npoints) {
            // 获取下一个呗
            cur_scalar_ptr += 1;
            cur_scalar = (uint32_t)(*(cur_scalar_ptr));
            // 获得之前处理的最低位
            cur_scalar += (scalar_tuple[j - npoints] & 1);
            scalar_tuple[j] = d_scalar_map[cur_scalar];

            point_idx[j] = i;
        }
    }

}
// 把输进来的scalar看作是u16
// 只支持窗口大小为 16 的...
__global__
void jy_process_scalar_1(uint16_t* scalar, uint32_t* scalar_tuple,
                      uint32_t* point_idx, size_t npoints) {

    const uint32_t tnum = blockDim.x * gridDim.x;
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // 每个线程分配到一个标量的划分
    for (int i = tid; i < npoints; i += tnum) {
        // 因为把他看作是u16 因此偏移需要加 2^16 * i
        // 当前线程处理的起始标量 ki0
        uint16_t* cur_scalar_ptr = scalar + (i << 4);
        // 获得标量值
        uint16_t cur_scalar = *cur_scalar_ptr;  // uint32_t instead of uint16_t, specifically for 0x10000
        uint32_t cur_sign = (cur_scalar >> (WBITS - 1)) & 1;
        cur_scalar = cur_sign == 1 ? 1<<WBITS - cur_scalar : cur_scalar;
        scalar_tuple[i] = cur_scalar << 1 | cur_sign;

        point_idx[i] = i;
        // j 放进去
        for (int j = i + npoints; j < NWINS * npoints; j += npoints) {
            // 获取下一个呗
            cur_scalar_ptr += 1;
            uint16_t cur_scalar = *cur_scalar_ptr;
            // 获得之前处理的最低位
            cur_scalar += (scalar_tuple[j - npoints] & 1);
            uint32_t cur_sign = (cur_scalar >> (WBITS - 1)) & 1;
            cur_scalar = cur_sign == 1 ? 1<<WBITS - cur_scalar : cur_scalar;
            point_idx[j] = i;
            scalar_tuple[i] = cur_scalar << 1 | cur_sign;
        }
    }

}
// bucket_idx_ptr 第i个窗口第j个值对应的就是排序后scalar的值
__global__
void process_scalar_2(uint32_t* scalar_tuple_out,
                      uint16_t* bucket_idx, size_t npoints) {
    // dim3(NWINS, config.N), NTHREADS
    // 线程总数
    // blockDim.x 每个 block 中的线程数量 NTHREADS
    // gridDim.y  grid 在 y 维度上的大小 config.N
    // 每加一个 tnum  线程块处理的总数
    const uint32_t tnum = blockDim.x * gridDim.y;
    // 全局索引
    // blockIdx.y 当前线程所在线程块在 y 方向哪里
    // blockDim.x 每个 block 中的线程数量 NTHREADS
    // 当前窗口的第 tid 个线程
    const uint32_t tid = blockIdx.y * blockDim.x + threadIdx.x;
    // 当前线程在 block x 方向上哪里  其实就是第几个 windows
    const uint32_t bid = blockIdx.x;

    // 当前线程处理第几个 windows
    // 窗口内对应的桶 idx 和排序处理后的 scalars
    uint16_t* bucket_idx_ptr = bucket_idx + npoints * bid;
    uint32_t* scalar_tuple_out_ptr = scalar_tuple_out + npoints * bid;
    // 每个线程处理当前窗口内负责的任务
    for (uint32_t i = tid; i < npoints; i += tnum) {
        // 取前 16 位
        // 桶索引对应的值是相应的scalar的前16位
        // 反正就是
        bucket_idx_ptr[i] = scalar_tuple_out_ptr[i] >> 16;
    }
}

// total_bucket_num = NWINS * (1 << (WBITS - 2))
__global__
void bucket_inf(bucket_t *buckets) {
    const uint32_t tnum = blockDim.x * gridDim.y;
    const uint32_t tid = blockIdx.y * blockDim.x + threadIdx.x;
    const uint32_t bid = blockIdx.x;

    const uint32_t bucket_num =  1 << (WBITS - 2);
    bucket_t* buckets_ptr = buckets + bucket_num * bid;

    for (uint32_t i = tid; i < bucket_num; i += tnum) {
        buckets_ptr[i].inf();
    }
}

// v1.1
__global__
void bucket_acc(uint32_t* scalar_tuple_out, uint16_t* bucket_idx, uint32_t* point_idx_out,
                affine_t* pre_points, bucket_t *buckets_pre,
                uint16_t* bucket_idx_pre_vector, uint16_t* bucket_idx_pre_used,
                uint32_t* bucket_idx_pre_offset, size_t npoints) {
    // 线程总数
    // 每增加一个 tnum 线程负责的任务
    const uint32_t tnum = blockDim.x * gridDim.y;
    // 线程块内部的一个 idx
    const uint32_t tid_inner = threadIdx.x;
    // 当前窗口的第 tid 个线程
    const uint32_t tid = blockIdx.y * blockDim.x + tid_inner;
    // 第 bid 个窗口 0 to NWINS - 1
    const uint32_t bid = blockIdx.x;
    // tnum + 2 ^(WBITS-2)
    const uint32_t buffer_len = tnum + (1 << (WBITS - 2));
    // 第 bid 个窗口对应的scalar标量
    uint32_t* scalar_tuple_out_ptr = scalar_tuple_out + npoints * bid;
    // 第 bid 个窗口对应的 bucket_idx
    uint16_t* bucket_idx_ptr = bucket_idx + npoints * bid;
    // 第 bid 个窗口对应的 点索引
    uint32_t* point_idx_out_ptr = point_idx_out + npoints * bid;
    // 和负载平衡相关
    // 只使用一个config.N * NTHREADS 个线程 处理每个窗口
    // 很明显，每个窗口分配的buffer是 buffer_len
    // 第 bid 个窗口对应的 bucket_pre
    bucket_t* buckets_pre_ptr = buckets_pre + buffer_len * bid;
    // 第 bid 个窗口对应的 bucket_index
    uint16_t* bucket_idx_pre_vector_ptr = bucket_idx_pre_vector + buffer_len * bid;
    // 第 bid 个窗口对应的 bucket_used
    uint16_t* bucket_idx_pre_used_ptr = bucket_idx_pre_used + tnum * bid;
    // 第 bid 个窗口对应的 bucket_offset
    uint32_t* bucket_idx_pre_offset_ptr = bucket_idx_pre_offset + tnum * bid;

    // 每个线程分配的任务 总数是tnum
    // 每个窗口内 每个线程处理的点数
    const uint32_t step_len = (npoints + tnum - 1) / tnum;
    // 首先确定边界范围，当然需要进一步调整
    uint32_t s = step_len * tid;
    uint32_t e = s + step_len;
    if (s >= npoints) {
        bucket_idx_pre_used_ptr[tid] = 0;
        return;
    }
    if (e >= npoints) e = npoints;

    uint16_t pre_bucket_idx = 0x8000;   // not exist
    // 线程块内部共享内存
    bucket_acc_smem[tid_inner * 2 + 1].inf(); // 设置为inf

    // 根据 scalar 值获得 offset
    // 第 s 个 bucket_idx 其实就是scalar i,j_bar
    // salar 是 odd 因此它对应的桶是 + 1 / 2
    uint32_t offset = tid + ((bucket_idx_ptr[s] + 1) >> 1);
    bucket_idx_pre_offset_ptr[tid] = offset;
    uint32_t unique_num = 0;
    // 每个线程在每个窗口下处理的点
    // process [s, e)
    for (uint32_t i = s; i < e; i++) {
        // todo 感觉就是在格式里面嵌入了新东西
        // 获得指数和剩余dbl次数
        uint32_t power_of_2 = (scalar_tuple_out_ptr[i] >> 8) & 0x0f;
        uint32_t dbl_time = (scalar_tuple_out_ptr[i] >> 12) & 0x0f;

        // 当前桶索引 其实就是 ai.ODD
        uint16_t cur_bucket_idx = bucket_idx_ptr[i];

        if (cur_bucket_idx != pre_bucket_idx && (unique_num++)) {
            // 因为unique_num ++ 了 索引就是 i != s 的时候 ,unique_num 起步等于2
            buckets_pre_ptr[offset + unique_num - 2] = bucket_acc_smem[tid_inner * 2 + 1];
            bucket_idx_pre_vector_ptr[offset + unique_num - 2] = (pre_bucket_idx + 1) >> 1;
            bucket_acc_smem[tid_inner * 2 + 1].inf();
        }
        pre_bucket_idx = cur_bucket_idx;
        // 查预计算表获得点值
        bucket_acc_smem[tid_inner * 2] = pre_points[point_idx_out_ptr[i] + power_of_2 * npoints];
        for (uint32_t j = 0; j < dbl_time; j++) {
            bucket_acc_smem[tid_inner * 2].dbl();
        }
        // 根据scalar的符号判断是否需要进行取反
        if (scalar_tuple_out_ptr[i] & 0x01) {
            bucket_acc_smem[tid_inner * 2].neg(true);
        }
        bucket_acc_smem[tid_inner * 2 + 1].add(bucket_acc_smem[tid_inner * 2]);
    }
    buckets_pre_ptr[offset + unique_num - 1] = bucket_acc_smem[tid_inner * 2 + 1];
    bucket_idx_pre_vector_ptr[offset + unique_num - 1] = (pre_bucket_idx + 1) >> 1;
    bucket_idx_pre_used_ptr[tid] = unique_num;

}

// v1.1 (2^{14} THREADS)
// 利用二分搜索去找相应的buffer点进行聚合到相应桶里
__global__
void bucket_acc_2(bucket_t *buckets_pre, uint16_t* bucket_idx_pre_vector, uint16_t* bucket_idx_pre_used,
                  uint32_t* bucket_idx_pre_offset, bucket_t *buckets, uint32_t upper_tnum, size_t npoints) {
    const uint32_t tid_inner = threadIdx.x;
    const uint32_t tid = blockIdx.y * blockDim.x + tid_inner;
    const uint32_t bid = blockIdx.x;
    const uint32_t buffer_len = upper_tnum + (1 << (WBITS - 2));
    // dim3(NWINS, (1 << (WBITS - 2)) / NTHREADS), NTHREADS 不能直接求tnum了
    // upper_tnum = (uint32_t)(config.N * NTHREADS)
    bucket_t* buckets_pre_ptr = buckets_pre + buffer_len * bid;
    uint16_t* bucket_idx_pre_vector_ptr = bucket_idx_pre_vector + buffer_len * bid;
    uint16_t* bucket_idx_pre_used_ptr = bucket_idx_pre_used + upper_tnum * bid;
    uint32_t* bucket_idx_pre_offset_ptr = bucket_idx_pre_offset + upper_tnum * bid;
    bucket_t* buckets_ptr = buckets + (1 << (WBITS - 2)) * bid;

    // 在每个窗口内查线程总数干的东西
    int left = 0, right = upper_tnum - 1;
    bool not_inf = false;
    uint32_t start_pos = 0;
    while (left <= right) {
        int mid = left + ((right - left) >> 1);
        uint16_t vector_used = bucket_idx_pre_used_ptr[mid];
        if (!vector_used) {
            right = mid - 1;
        } else {
            uint32_t vector_ptr = bucket_idx_pre_offset_ptr[mid];
            uint16_t min_idx = bucket_idx_pre_vector_ptr[vector_ptr];
            uint16_t max_idx = bucket_idx_pre_vector_ptr[vector_ptr + vector_used - 1];
            if (min_idx == (tid + 1)) {
                start_pos = mid;
                not_inf = true;
                right = mid - 1;
            } else if (min_idx > (tid + 1)) {
                right = mid - 1;
            } else if (max_idx < (tid + 1)) {
                left = mid + 1;
            } else {
                for (uint32_t i = vector_ptr + 1; i < vector_ptr + vector_used; i++) {
                    if (bucket_idx_pre_vector_ptr[i] == (tid + 1)) {
                        start_pos = mid;
                        not_inf = true;
                        break;
                    }
                }
                break;
            }
        }
    }
    bucket_acc_smem[tid_inner].inf();
    while (not_inf && start_pos < upper_tnum) {
        not_inf = false;
        // 找到对应的buffer了
        uint16_t vector_used = bucket_idx_pre_used_ptr[start_pos];
        uint32_t vector_ptr = bucket_idx_pre_offset_ptr[start_pos];
        for (uint32_t i = vector_ptr; i < vector_ptr + vector_used; i++) {
            if (bucket_idx_pre_vector_ptr[i] == (tid + 1)) {
                not_inf = true;
                // 把找到的点累加起来
                bucket_acc_smem[tid_inner].add(buckets_pre_ptr[i]);
                break;
            }
        }
        // 然后往前找
	    start_pos++;
    }
    // 最后存到相应的全局内存里
    buckets_ptr[tid] = bucket_acc_smem[tid_inner];  // can omit kerner `bucket_inf`

}

__global__
void bucket_agg_1(bucket_t *buckets) {
    // dim3(NWINS, config.N), NTHREADS
    const uint32_t tnum = blockDim.x * gridDim.y;
    const uint32_t tid = blockIdx.y * blockDim.x + threadIdx.x;
    const uint32_t bid = blockIdx.x;

    // 第 i 个窗口对应的桶值
    bucket_t* buckets_ptr = buckets + (1 << (WBITS - 2)) * bid;

    for (uint32_t j = tid; j < (1 << (WBITS - 5)); j += tnum) {
        uint32_t s = j << 3;
        bucket_t* Bi = buckets_ptr + 0x3fff - s;
        for (int i = 1; i < 8; i++) {
            (Bi - i)->add(*(Bi - i + 1));
        }
    }
}

__global__
void bucket_agg_2(bucket_t *buckets) {
    const uint32_t tnum = blockDim.x * gridDim.y;
    const uint32_t tid = blockIdx.y * blockDim.x + threadIdx.x;
    const uint32_t bid = blockIdx.x;

    bucket_t* buckets_ptr = buckets + (1 << (WBITS - 2)) * bid;

    for (uint32_t i = 3; i < 14; i++) {
        for (uint32_t k = tid; k < (1 << (WBITS - 3)); k += tnum) {
            uint32_t baseline = ((1 + (k >> i)) << (i + 1)) - (1 << i);
            uint32_t offset = k & ((1 << i) - 1);

            bucket_t* Bi = buckets_ptr + 0x3fff - (baseline - 1);
            bucket_t* Bj = Bi - (offset + 1);	// B + 0x3fff - (baseline + offset)

            Bj->add(*Bi);
        }
        cooperative_groups::this_grid().sync();
    }
}

__global__
void recursive_sum(bucket_t *buckets, bucket_t *res) {
    // dim3(NWINS, config.N), NTHREADS
    // res 为每个窗口对应的桶和 即算Qj
    const uint32_t tnum = blockDim.x * gridDim.y;
    const uint32_t tid = blockIdx.y * blockDim.x + threadIdx.x;
    const uint32_t bid = blockIdx.x;

    bucket_t* buckets_ptr = buckets + (1 << (WBITS - 2)) * bid;

    if (tid == 0) {
        res[bid] = *buckets_ptr;
    }
    // cooperative_groups::this_grid().sync();

    for (uint32_t j = 1 << (WBITS - 3); j > NTHREADS; j >>= 1) {
        for (uint32_t i = tid; i < j; i += tnum) {
            buckets_ptr[i].add(buckets_ptr[i + j]);
        }
        cooperative_groups::this_grid().sync();
    }
    for (uint32_t j = NTHREADS; j > WARP_SZ; j >>= 1) {
        if (tid < j) {
            buckets_ptr[tid].add(buckets_ptr[tid + j]);
        }
        cooperative_groups::this_thread_block().sync();
    }

    if (tid < WARP_SZ) {
        buckets_ptr[tid].add(buckets_ptr[tid + 32]);
        buckets_ptr[tid].add(buckets_ptr[tid + 16]);
        buckets_ptr[tid].add(buckets_ptr[tid + 8]);
        buckets_ptr[tid].add(buckets_ptr[tid + 4]);
        buckets_ptr[tid].add(buckets_ptr[tid + 2]);
        buckets_ptr[tid].add(buckets_ptr[tid + 1]);
    }
    if (tid == 0) {
        // 2Qj
        buckets_ptr->dbl();
        // -B1
        res[bid].neg(true);
        // 2Qj-B1
        res[bid].add(*buckets_ptr);
    }

    /*cooperative_groups::this_grid().sync();
    if (tid == 0 && bid == 0) {
    bucket_t check_res;
    check_res.inf();

    for (int i = 15; i > -1; i--) {
	for (int j = 0; j < 16; j++) {
	    check_res.add(check_res);
	}
	check_res.add(res[i]);
    }
    printf("\ncheck_2:\n");
    check_res.xyzz_print();
    }*/
}

#else

#include <cassert>
#include <vector>
using namespace std;

#include <util/exception.cuh>
#include <util/rusterror.h>
#include <util/thread_pool_t.hpp>
#include <util/host_pinned_allocator_t.hpp>


template<typename... Types>
inline void launch_coop(void(*f)(Types...),
                        dim3 gridDim, dim3 blockDim, hipStream_t stream,
                        Types... args)
{
    void* va_args[sizeof...(args)] = { &args... };
    CUDA_OK(hipLaunchCooperativeKernel((const void*)f, gridDim, blockDim,
                                        va_args, 0, stream));
}

class stream_t {
    hipStream_t stream;
public:
    stream_t(int device)  {
        CUDA_OK(hipSetDevice(device));
        hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    }
    ~stream_t() { hipStreamDestroy(stream); }
    inline operator decltype(stream)() { return stream; }
};


template<class bucket_t> class result_t_faster {
    bucket_t ret[NWINS];
public:
    result_t_faster() {}
    inline operator decltype(ret)&() { return ret; }
};

template<class T>
class device_ptr_list_t {
    vector<T*> d_ptrs;
public:
    device_ptr_list_t() {}
    ~device_ptr_list_t() {
        for(T *ptr: d_ptrs) {
            hipFree(ptr);
        }
    }
    size_t allocate(size_t bytes) {
        T *d_ptr;
        CUDA_OK(hipMalloc(&d_ptr, bytes));
        d_ptrs.push_back(d_ptr);
        return d_ptrs.size() - 1;
    }
    size_t size() {
        return d_ptrs.size();
    }
    T* operator[](size_t i) {
        if (i > d_ptrs.size() - 1) {
            CUDA_OK(hipErrorInvalidDevicePointer);
        }
        return d_ptrs[i];
    }

};

// Pippenger MSM class
template<class bucket_t, class point_t, class affine_t, class scalar_t>
class pippenger_t {
public:
    typedef vector<result_t_faster<bucket_t>,
                   host_pinned_allocator_t<result_t_faster<bucket_t>>> result_container_t_faster;

private:
    size_t sm_count;
    bool init_done = false;
    device_ptr_list_t<affine_t> d_base_ptrs;
    // 预计算点
    device_ptr_list_t<affine_t> d_pre_points_ptrs;
    device_ptr_list_t<scalar_t> d_scalar_ptrs;
    device_ptr_list_t<bucket_t> d_bucket_ptrs;

    device_ptr_list_t<bucket_t> d_bucket_pre_ptrs;  // v1.1
    device_ptr_list_t<uint16_t> d_bucket_idx_pre_ptrs;  // v1.1
    device_ptr_list_t<uint32_t> d_bucket_idx_pre2_ptrs;  // v1.2

    device_ptr_list_t<bucket_t> d_res_ptrs;

    // GPU device number
    int device;

    // TODO: Move to device class eventually
    thread_pool_t *da_pool = nullptr;

public:
    // Default stream for operations
    stream_t default_stream;

    device_ptr_list_t<uint32_t> d_scalar_map;
    device_ptr_list_t<uint32_t> d_scalar_tuple_ptrs;
    device_ptr_list_t<uint32_t> d_point_idx_ptrs;
    // 符号变换
    device_ptr_list_t<uint32_t> jy_d_scalar_tuple_ptrs;
    device_ptr_list_t<uint32_t> jy_d_point_idx_ptrs;


    device_ptr_list_t<uint16_t> d_bucket_idx_ptrs;
    device_ptr_list_t<unsigned char> d_cub_ptrs;

    // Parameters for an MSM operation
    class MSMConfig {
        friend pippenger_t;
    public:
        size_t npoints;
        size_t N;
        size_t n;
    };

    pippenger_t() : default_stream(0) {
        device = 0;
    }

    pippenger_t(int _device, thread_pool_t *pool = nullptr)
        : default_stream(_device) {
        da_pool = pool;
        device = _device;
    }

    // Initialize instance. Throws cuda_error on error.
    void init() {
        printf("[Initialize GPU instance.]\n");
        if (!init_done) {
            CUDA_OK(hipSetDevice(device));
            hipDeviceProp_t prop;
            if (hipGetDeviceProperties(&prop, 0) != hipSuccess || prop.major < 7)
                CUDA_OK(hipErrorInvalidDevice);
            sm_count = prop.multiProcessorCount;

            if (da_pool == nullptr) {
                da_pool = new thread_pool_t();
            }

            init_done = true;
        }
    }

    int get_device() {
        return device;
    }

    // Initialize parameters for a specific size MSM. Throws cuda_error on error.
    MSMConfig init_msm_faster(size_t npoints) {
        printf("[Begin init MSMConfig parameters]\n");
        init();

        MSMConfig config;
        config.npoints = npoints;
        config.n = (npoints+WARP_SZ-1) & ((size_t)0-WARP_SZ);
        // todo 可能需要修改
        config.N = (sm_count*256) / (NTHREADS*NWINS);
        size_t delta = ((npoints+(config.N)-1)/(config.N)+WARP_SZ-1) & (0U-WARP_SZ);
        config.N = (npoints+delta-1) / delta;
        printf("[MSMConfig] [npoints] [%d] [config.n] [%d] [delta] [%d] [Config.N] [%d]\n", npoints, config.n, delta, config.N);

        //        if(config.N % 2 == 1) config.N -= 1;
        return config;
    }

    size_t get_size_bases(MSMConfig& config) {
        return config.n * sizeof(affine_t);
    }
    size_t get_size_scalars(MSMConfig& config) {
        return config.n * sizeof(scalar_t);
    }
    // 窗口数乘以 2 ^ c - 2
    size_t get_size_buckets() {
        return sizeof(bucket_t) * NWINS * (1 << (WBITS - 2));
    }
    size_t get_size_buckets_pre(MSMConfig& config) { // v1.1
        return sizeof(bucket_t) * NWINS * (config.N * NTHREADS + (1 << (WBITS - 2)));
    }
    size_t get_size_bucket_idx_pre_vector(MSMConfig& config) {  // v1.1
        return sizeof(uint16_t) * NWINS * (config.N * NTHREADS + (1 << (WBITS - 2)));
    }
    size_t get_size_bucket_idx_pre_used(MSMConfig& config) {  // v1.1
        return sizeof(uint16_t) * config.N * NTHREADS * NWINS;
    }
    size_t get_size_bucket_idx_pre_offset(MSMConfig& config) {  // v1.2
        return sizeof(uint32_t) * config.N * NTHREADS * NWINS;
    }
    // 窗口数 * 桶大小
    size_t get_size_res() {
        return sizeof(bucket_t) * NWINS;
    }
    // (2^c + 1) * kij 的组合形式
    size_t get_size_scalar_map() {
        return ((1 << 16) + 1) * sizeof(uint32_t);
    }
    // scalar tuple 存放 kij  uint32 * NWINS * 点数
    size_t get_size_scalar_tuple(MSMConfig& config) {
        return config.n * sizeof(uint32_t) * NWINS;
    }
    size_t get_size_point_idx(MSMConfig& config) {
        return config.n * sizeof(uint32_t) * NWINS;
    }
    // 桶索引大小 点数 * 窗口数 * 窗口内桶索引
    size_t get_size_bucket_idx(MSMConfig& config) {
        return config.n * sizeof(uint16_t) * NWINS;
    }
    // 分配 cub 排序所需空间
    size_t get_size_cub_sort_faster(MSMConfig& config){
        uint32_t *d_scalar_tuple = nullptr;
        uint32_t *d_scalar_tuple_out = nullptr;
        uint32_t *d_point_idx = nullptr;
        uint32_t *d_point_idx_out = nullptr;
        void *d_temp = NULL;
        size_t temp_size = 0;
        hipcub::DeviceRadixSort::SortPairs(d_temp, temp_size,
                                        d_scalar_tuple, d_scalar_tuple_out,
                                        d_point_idx, d_point_idx_out, config.n, 0, 31);
        return temp_size;
    }

    result_container_t_faster get_result_container_faster() {
        result_container_t_faster res(1);
        return res;
    }

    // Allocate storage for bases on device. Throws cuda_error on error.
    // Returns index of the allocated base storage.
    // 7 是 原 points + 预计算的 六 组点
    size_t allocate_d_bases(MSMConfig& config) {
        printf("[Allocate d_bases] 7 * config.n * sizeof(affine_t) [%d]\n",7 * get_size_bases(config));
        return d_base_ptrs.allocate(7 * get_size_bases(config));
    }

    size_t allocate_d_pre_points(MSMConfig& config) {
        // 11 个窗口 => 2^2c  2^4c 2^6c 2^8c 2&10c  + 原来那组
        size_t num = (NWINS % 2 ==0 ? NWINS - 2 : NWINS - 1) / 2 + 1;
        return d_pre_points_ptrs.allocate( num * get_size_bases(config));
    }

    size_t allocate_d_scalars(MSMConfig& config) {
        printf("[Allocate d_scalars] config.n * sizeof(scalar_t) [%d]\n",get_size_scalars(config));
        return d_scalar_ptrs.allocate(get_size_scalars(config));
    }

    size_t allocate_d_buckets() {
        printf("[Allocate d_buckets] sizeof(bucket_t) * NWINS * (1 << (WBITS - 2)) [%d]\n",get_size_buckets());
        return d_bucket_ptrs.allocate(get_size_buckets());
    }
    // 静态 bucket
    size_t allocate_d_buckets_pre(MSMConfig& config) {  // v1.1
        printf("[Allocate d_buckets_pre] sizeof(bucket_t) * NWINS * (config.N * NTHREADS + (1 << (WBITS - 2))) [%d]\n",get_size_buckets_pre(config));
        return d_bucket_pre_ptrs.allocate(get_size_buckets_pre(config));
    }
    // buffer_index
    size_t allocate_d_bucket_idx_pre_vector(MSMConfig& config) {  // v1.1
        printf("[Allocate d_bucket_idx_pre_vector] sizeof(uint16_t) * NWINS * (config.N * NTHREADS + (1 << (WBITS - 2))) [%d]\n",get_size_bucket_idx_pre_vector(config));
        return d_bucket_idx_pre_ptrs.allocate(get_size_bucket_idx_pre_vector(config));
    }
    // buffer_used
    size_t allocate_d_bucket_idx_pre_used(MSMConfig& config) {  // v1.1
        printf("[Allocate d_bucket_idx_pre_used] sizeof(uint16_t) * config.N * NTHREADS * NWINS [%d]\n",get_size_bucket_idx_pre_used(config));
        return d_bucket_idx_pre_ptrs.allocate(get_size_bucket_idx_pre_used(config));
    }
    // buffer_offset
    size_t allocate_d_bucket_idx_pre_offset(MSMConfig& config) {  // v1.2
        printf("[Allocate d_bucket_idx_pre_offset] sizeof(uint32_t) * config.N * NTHREADS * NWINS [%d]\n",get_size_bucket_idx_pre_offset(config));
        return d_bucket_idx_pre2_ptrs.allocate(get_size_bucket_idx_pre_offset(config));
    }

    size_t allocate_d_res() {
        printf("[Allocate d_res] sizeof(bucket_t) * NWINS [%d]\n",get_size_res());
        return d_res_ptrs.allocate(get_size_res());
    }

    size_t allocate_d_scalar_map() {
        printf("[Allocate d_scalar_map] ((1 << 16) + 1) * sizeof(uint32_t) [%d]\n",get_size_scalar_map());
        return d_scalar_map.allocate(get_size_scalar_map());
    }

    size_t allocate_jy_d_scalar_tuple(MSMConfig& config) {
        return jy_d_scalar_tuple_ptrs.allocate(get_size_scalar_tuple(config));
    }
    size_t allocate_jy_d_scalar_tuple_out(MSMConfig& config) {
        return jy_d_scalar_tuple_ptrs.allocate(get_size_scalar_tuple(config));
    }
    size_t allocate_jy_d_point_idx(MSMConfig& config) {
        return jy_d_point_idx_ptrs.allocate(get_size_point_idx(config));
    }
    size_t allocate_jy_d_point_idx_out(MSMConfig& config) {
        return jy_d_point_idx_ptrs.allocate(get_size_point_idx(config));
    }
    size_t allocate_d_scalar_tuple(MSMConfig& config) {
        printf("[Allocate d_scalar_tuple] config.n * sizeof(uint32_t) * NWINS [%d]\n",get_size_scalar_tuple(config));
        return d_scalar_tuple_ptrs.allocate(get_size_scalar_tuple(config));
    }
    size_t allocate_d_scalar_tuple_out(MSMConfig& config) {
        printf("[Allocate d_scalar_tuple_out] config.n * sizeof(uint32_t) * NWINS [%d]\n",get_size_scalar_tuple(config));
        return d_scalar_tuple_ptrs.allocate(get_size_scalar_tuple(config));
    }

    size_t allocate_d_point_idx(MSMConfig& config) {
        printf("[Allocate d_point_idx] config.n * sizeof(uint32_t) * NWINS [%d]\n",get_size_point_idx(config));
        return d_point_idx_ptrs.allocate(get_size_point_idx(config));
//        return d_point_idx_ptrs.allocate(config.n * sizeof(uint32_t));
    }
    size_t allocate_d_point_idx_out(MSMConfig& config) {
        printf("[Allocate d_point_idx_out] config.n * sizeof(uint32_t) * NWINS [%d]\n",get_size_point_idx(config));
        return d_point_idx_ptrs.allocate(get_size_point_idx(config));
    }
    // 分配桶索引空间
    size_t allocate_d_bucket_idx(MSMConfig& config) {
        printf("[Allocate d_bucket_idx] config.n * sizeof(uint16_t) * NWINS [%d]\n",get_size_bucket_idx(config));
        return d_bucket_idx_ptrs.allocate(get_size_bucket_idx(config));
    }

    size_t allocate_d_cub_sort_faster(MSMConfig& config) {
        printf("[Allocate d_cub_sort_faster WARN Change] config.n * sizeof(uint16_t) * NWINS [%d]\n",get_size_cub_sort_faster(config));
        return d_cub_ptrs.allocate(get_size_cub_sort_faster(config));
    }

    // Transfer bases to device. Throws cuda_error on error.
    void transfer_bases_to_device(MSMConfig& config, size_t d_bases_idx, const affine_t points[],
                                  size_t ffi_affine_sz = sizeof(affine_t),
                                  hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        affine_t *d_points = d_base_ptrs[d_bases_idx];
        CUDA_OK(hipSetDevice(device));
        if (ffi_affine_sz != sizeof(*d_points))
            CUDA_OK(hipMemcpy2DAsync(d_points, sizeof(*d_points),
                                      points, ffi_affine_sz,
                                      ffi_affine_sz, config.npoints,
                                      hipMemcpyHostToDevice, stream));
        else
            CUDA_OK(hipMemcpyAsync(d_points, points, config.npoints*sizeof(*d_points),
                                    hipMemcpyHostToDevice, stream));
    }

    // Transfer scalars to device. Throws cuda_error on error.
    void transfer_scalars_to_device(MSMConfig& config,
                                    size_t d_scalars_idx, const scalar_t scalars[],
                                    hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        scalar_t *d_scalars = d_scalar_ptrs[d_scalars_idx];
        CUDA_OK(hipSetDevice(device));
        CUDA_OK(hipMemcpyAsync(d_scalars, scalars, config.npoints*sizeof(*d_scalars),
                                hipMemcpyHostToDevice, stream));
    }


    void transfer_res_to_host_faster(result_container_t_faster &res, size_t d_res_idx,
                                  hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        bucket_t *d_res = d_res_ptrs[d_res_idx];
        CUDA_OK(hipSetDevice(device));
        CUDA_OK(hipMemcpyAsync(res[0], d_res, sizeof(res[0]),
                                hipMemcpyDeviceToHost, stream));
    }

    void transfer_scalar_map_to_device(size_t d_scalar_map_idx, const uint32_t scalar_map[],
                                       hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        uint32_t *d_smap = d_scalar_map[d_scalar_map_idx];
        CUDA_OK(hipSetDevice(device));
        CUDA_OK(hipMemcpyAsync(d_smap, scalar_map, ((1 << 16) + 1)*sizeof(uint32_t),
                                hipMemcpyHostToDevice, stream));
    }

    void synchronize_stream() {
        CUDA_OK(hipSetDevice(device));
        CUDA_OK(hipStreamSynchronize(default_stream));
    }

    void launch_kernel_init(MSMConfig& config,
                            size_t d_points_sn, hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        affine_t *d_points = d_base_ptrs[d_points_sn];

        CUDA_OK(hipSetDevice(device));
        printf("[pre_compute] NWINS * config.N  NTHREADS \n");
        launch_coop(pre_compute, NWINS * config.N, NTHREADS, stream,
                    d_points, config.npoints);
    }

    void launch_kernel_pre_compute_init(MSMConfig& config,
                                        size_t d_pre_points_sn, hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        affine_t *d_pre_points = d_pre_points_ptrs[d_pre_points_sn];

        CUDA_OK(hipSetDevice(device));
        launch_coop(jy_pre_compute, NWINS * config.N, NTHREADS, stream,
                    d_pre_points, config.npoints);
    }
    // conversion of the sub-scalars (table lookups).
    // d_scalars_sn 标量地址
    // d_scalar_tuples_sn 标量元组地址
    // 查找表地址
    // 点索引地址
    void launch_process_scalar_1(MSMConfig& config,
                                 size_t d_scalars_sn, size_t d_scalar_tuples_sn,
                                 size_t d_scalar_map_sn, size_t d_point_idx_sn,
                                 hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        uint16_t* d_scalars = (uint16_t*)d_scalar_ptrs[d_scalars_sn];
        uint32_t* d_scalar_tuple = d_scalar_tuple_ptrs[d_scalar_tuples_sn];
        uint32_t* d_smap = d_scalar_map[d_scalar_map_sn];
        uint32_t* d_point_idx = d_point_idx_ptrs[d_point_idx_sn];

        CUDA_OK(hipSetDevice(device));
        launch_coop(process_scalar_1, NWINS * config.N, NTHREADS, stream,
                    d_scalars, d_scalar_tuple, d_smap, d_point_idx, config.npoints);
    }

    void launch_jy_process_scalar_1(MSMConfig& config,
                                 size_t d_scalars_sn, size_t jy_d_scalar_tuples_sn,
                                 size_t jy_d_point_idx_sn,
                                 hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        uint16_t* d_scalars = (uint16_t*)d_scalar_ptrs[d_scalars_sn];
        uint32_t* d_scalar_tuple = jy_d_scalar_tuple_ptrs[jy_d_scalar_tuples_sn];
        uint32_t* d_point_idx = jy_d_point_idx_ptrs[jy_d_point_idx_sn];

        CUDA_OK(hipSetDevice(device));
        launch_coop(jy_process_scalar_1, NWINS * config.N, NTHREADS, stream,
                    d_scalars, d_scalar_tuple, d_point_idx, config.npoints);
    }

    // 根据排序后的scalar元组，获得桶idx
    void launch_process_scalar_2(MSMConfig& config,
                                 size_t d_scalar_tuples_out_sn, size_t d_bucket_idx_sn,
                                 hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        uint32_t* d_scalar_tuple_out = d_scalar_tuple_ptrs[d_scalar_tuples_out_sn];
        uint16_t* d_bucket_idx = d_bucket_idx_ptrs[d_bucket_idx_sn];

        CUDA_OK(hipSetDevice(device));
        // NWINS 是网格在 x 维度上的大小，config.N 是网格在 y 维度上的大小。
        // 看成是二维的即可
        launch_coop(process_scalar_2, dim3(NWINS, config.N), NTHREADS, stream,
                    d_scalar_tuple_out, d_bucket_idx, config.npoints);
    }

    void launch_bucket_inf(MSMConfig& config, size_t d_buckets_sn, hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        bucket_t* d_buckets = d_bucket_ptrs[d_buckets_sn];

        CUDA_OK(hipSetDevice(device));
        launch_coop(bucket_inf, dim3(NWINS, config.N), NTHREADS, stream, d_buckets);
    }

    void launch_bucket_acc(MSMConfig& config,
                           size_t d_scalar_tuples_out_sn, size_t d_bucket_idx_sn,
                           size_t d_point_idx_out_sn, size_t d_points_sn, size_t d_buckets_sn,
                           size_t d_buckets_pre_sn, size_t d_bucket_idx_pre_vector_sn,
                           size_t d_bucket_idx_pre_used_sn, size_t d_bucket_idx_pre_offset_sn,
                           hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        uint32_t* d_scalar_tuple_out = d_scalar_tuple_ptrs[d_scalar_tuples_out_sn];
        uint16_t* d_bucket_idx = d_bucket_idx_ptrs[d_bucket_idx_sn];
        uint32_t* d_point_idx_out = d_point_idx_ptrs[d_point_idx_out_sn];
        affine_t* d_points = d_base_ptrs[d_points_sn];
        bucket_t* d_buckets = d_bucket_ptrs[d_buckets_sn];
        bucket_t* d_buckets_pre = d_bucket_pre_ptrs[d_buckets_pre_sn];
        uint16_t* d_bucket_idx_pre_vector = d_bucket_idx_pre_ptrs[d_bucket_idx_pre_vector_sn];
        uint16_t* d_bucket_idx_pre_used = d_bucket_idx_pre_ptrs[d_bucket_idx_pre_used_sn];
        uint32_t* d_bucket_idx_pre_offset = d_bucket_idx_pre2_ptrs[d_bucket_idx_pre_offset_sn];

        CUDA_OK(hipSetDevice(device));
        // accumulate parts of the buckets into static buffers.
        launch_coop(bucket_acc, dim3(NWINS, config.N), NTHREADS, stream,
                    d_scalar_tuple_out, d_bucket_idx, d_point_idx_out,
                    d_points, d_buckets_pre,
                    d_bucket_idx_pre_vector, d_bucket_idx_pre_used,
                    d_bucket_idx_pre_offset, config.npoints);
        // aggregate the buffered points into the buckets.
        bucket_acc_2<<<dim3(NWINS, (1 << (WBITS - 2)) / NTHREADS), NTHREADS, 0, stream>>>(
                d_buckets_pre, d_bucket_idx_pre_vector, d_bucket_idx_pre_used,
                d_bucket_idx_pre_offset, d_buckets, (uint32_t)(config.N * NTHREADS), config.npoints
                );
//        launch_coop(bucket_acc_2, dim3(NWINS, (1 << (WBITS - 2)) / NTHREADS), NTHREADS, stream,
//                    d_buckets_pre, d_bucket_idx_pre_vector, d_bucket_idx_pre_used,
//                    d_bucket_idx_pre_offset, d_buckets, (uint32_t)(config.N * NTHREADS), config.npoints);

    }

    void launch_bucket_agg_1(MSMConfig& config, size_t d_buckets_sn, hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        bucket_t* d_buckets = d_bucket_ptrs[d_buckets_sn];

        CUDA_OK(hipSetDevice(device));
//        bucket_agg_1<<<dim3(NWINS, (1 << (WBITS - 5)) / NTHREADS), NTHREADS, 0, stream>>>(d_buckets);
        launch_coop(bucket_agg_1, dim3(NWINS, config.N), NTHREADS, stream, d_buckets);
    }

    void launch_bucket_agg_2(MSMConfig& config, size_t d_buckets_sn, hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        bucket_t* d_buckets = d_bucket_ptrs[d_buckets_sn];

        CUDA_OK(hipSetDevice(device));
        launch_coop(bucket_agg_2, dim3(NWINS, config.N), NTHREADS, stream, d_buckets);
    }

    void launch_recursive_sum(MSMConfig& config, size_t d_buckets_sn, size_t d_res_sn, hipStream_t s = nullptr) {
        hipStream_t stream = (s == nullptr) ? default_stream : s;
        bucket_t* d_buckets = d_bucket_ptrs[d_buckets_sn];
        bucket_t* d_res = d_res_ptrs[d_res_sn];

        CUDA_OK(hipSetDevice(device));
        launch_coop(recursive_sum, dim3(NWINS, config.N), NTHREADS, stream, d_buckets, d_res);
    }

    // Perform final accumulation on CPU.
    void accumulate_faster(point_t &out, result_container_t_faster &res) {
        out.inf();

        for(int32_t k = NWINS - 1; k >= 0; k--)
        {
            for (int32_t i = 0; i < WBITS; i++)
            {
                out.dbl();
            }
            point_t p = (res[0])[k];
            out.add(p);
        }

    }
};

#endif
