// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <hipcub/hipcub.hpp>

#include <ff/bls12-381.hpp>
#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_inf_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

// init cub radix sort
extern "C" void cub_initial()
{
    uint32_t *d_scalar_tuple = nullptr;
    uint32_t *d_scalar_tuple_out = nullptr;
    uint32_t *d_point_idx = nullptr;
    uint32_t *d_point_idx_out = nullptr;
    uint32_t *d_offset_a = nullptr;
    uint32_t *d_offset_b = nullptr;
    void *d_temp = NULL;
    size_t temp_size = 0;
    hipcub::DeviceRadixSort::SortPairs(d_temp, temp_size,
                                    d_scalar_tuple, d_scalar_tuple_out,
                                    d_point_idx, d_point_idx_out, 1);
    hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp, temp_size,
                                             d_scalar_tuple, d_scalar_tuple_out,
                                             d_point_idx, d_point_idx_out, 1, 1, d_offset_a, d_offset_b);
}

#ifndef __CUDA_ARCH__

// 每批次启动线程数
static const size_t NUM_BATCH_THREADS = 2;
static thread_pool_t batch_pool(NUM_BATCH_THREADS);

typedef pippenger_t<bucket_t, point_t, affine_t, scalar_t> pipp_t;

// MSM context used store persistent state
template <class bucket_t, class affine_t, class scalar_t>
struct Context
{
    // pippenger
    pipp_t pipp;
    // MSMConfig
    typename pipp_t::MSMConfig config;
    // 仿射点大小
    size_t ffi_affine_sz;
    // 预计算 point 包括 原始 point
    // p1 p2 p3 ... pn   2^2c p1  2^2c p2  ...  2^2c pn ....
    size_t d_pre_points_sn;
    // 批次数
    size_t d_scalars_sn[NUM_BATCH_THREADS];
    // 桶索引
    size_t d_buckets_sn;
    // 标量数组索引
    // k1,1 k2,1,...,kn,1   k1,2,k2,2 ,...kn,2  ...  k1,lambda/c...kn,lambda/c
    size_t jy_d_scalar_tuples_sn;
    // 标量对应点索引
    // p1,p2,p3,..,...pn   p1,p2,...,pn     ...      p1,p2,...,pn
    size_t jy_d_point_idx_sn;
    // 排序标量索引
    // 对每个窗口进行排序后的值
    size_t jy_d_scalar_tuples_out_sn;
    // 与之相对应的 point 索引
    size_t jy_d_point_idx_out_sn;
    // 用于负载平衡的 buffer
    // buffer
    size_t d_buckets_pre_sn;
    // buffer index
    size_t d_bucket_idx_pre_vector_sn;
    // buffer used
    size_t d_bucket_idx_pre_used_sn;
    // buffer offest
    size_t d_bucket_idx_pre_offset_sn;
    // res
    size_t d_res_sn;

    size_t d_st_sn;
    size_t d_sost_sn;
    // point => buffer index
    size_t d_bucket_idx_sn;
    // cub
    size_t d_cub_sort_idx;
    // host scalars
    scalar_t *h_scalars;

    typename pipp_t::result_container_t_faster fres0;
    typename pipp_t::result_container_t_faster fres1;
};

template <class bucket_t, class affine_t, class scalar_t>
struct RustContext
{
    Context<bucket_t, affine_t, scalar_t> *context;
};

// Initialization function
// Allocate device storage, transfer bases
extern "C" RustError mult_pippenger_faster_init(RustContext<bucket_t, affine_t, scalar_t> *context,
                                                const affine_t points[], size_t npoints,
                                                size_t ffi_affine_sz)
{
    context->context = new Context<bucket_t, affine_t, scalar_t>();
    Context<bucket_t, affine_t, scalar_t> *ctx = context->context;
    ctx->ffi_affine_sz = ffi_affine_sz;
    try
    {
        ctx->config = ctx->pipp.init_msm_faster(npoints);

        // Allocate GPU storage
        // 分配预计算点空间
        ctx->d_pre_points_sn = ctx->pipp.allocate_d_pre_points(ctx->config);
        //
        for (size_t i = 0; i < NUM_BATCH_THREADS; i++)
        {
            ctx->d_scalars_sn[i] = ctx->pipp.allocate_d_scalars(ctx->config);
        }
        // 分配桶空间
        ctx->d_buckets_sn = ctx->pipp.allocate_d_buckets();
        // 静态 bucket
        ctx->d_buckets_pre_sn = ctx->pipp.allocate_d_buckets_pre(ctx->config);
        // buffer_index
        ctx->d_bucket_idx_pre_vector_sn = ctx->pipp.allocate_d_bucket_idx_pre_vector(ctx->config);
        // buffer_used
        ctx->d_bucket_idx_pre_used_sn = ctx->pipp.allocate_d_bucket_idx_pre_used(ctx->config);
        // buffer_offset
        ctx->d_bucket_idx_pre_offset_sn = ctx->pipp.allocate_d_bucket_idx_pre_offset(ctx->config);

        ctx->d_st_sn = ctx->pipp.allocate_d_st(ctx->config);
        ctx->d_sost_sn = ctx->pipp.allocate_d_sost(ctx->config);
        // 返回值 NWIN * bucket
        ctx->d_res_sn = ctx->pipp.allocate_d_res();
        // 分配符号变换空间
        ctx->jy_d_scalar_tuples_sn = ctx->pipp.allocate_jy_d_scalar_tuple(ctx->config);
        ctx->jy_d_point_idx_sn = ctx->pipp.allocate_jy_d_point_idx(ctx->config);
        ctx->jy_d_scalar_tuples_out_sn = ctx->pipp.allocate_jy_d_scalar_tuple_out(ctx->config);
        ctx->jy_d_point_idx_out_sn = ctx->pipp.allocate_jy_d_point_idx(ctx->config);
        // 分配桶索引空间
        ctx->d_bucket_idx_sn = ctx->pipp.allocate_d_bucket_idx(ctx->config);
        // CUB 排序
        ctx->d_cub_sort_idx = ctx->pipp.allocate_d_cub_sort_faster(ctx->config);

        // Allocate pinned memory on host
        CUDA_OK(hipHostMalloc(&ctx->h_scalars, ctx->pipp.get_size_scalars(ctx->config)));

        // 传输到预计算点那组
        ctx->pipp.transfer_bases_to_device(ctx->config, ctx->d_pre_points_sn, points,
                                           ffi_affine_sz);
        ctx->pipp.launch_kernel_pre_compute_init(ctx->config, ctx->d_pre_points_sn);

        ctx->fres0 = ctx->pipp.get_result_container_faster();
        ctx->fres1 = ctx->pipp.get_result_container_faster();
    }
    catch (const cuda_error &e)
    {
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
        return RustError{e.code(), e.what()};
#else
        return RustError { e.code() }
#endif
    }
    return RustError{hipSuccess};
}

// Peform MSM on a batch of scalars over fixed bases
extern "C" RustError mult_pippenger_faster_inf(RustContext<bucket_t, affine_t, scalar_t> *context,
                                               point_t *out, const affine_t points[],
                                               size_t npoints, size_t batches,
                                               const scalar_t scalars[],
                                               size_t ffi_affine_sz)
{
    (void)points; // Silence unused param warning

    Context<bucket_t, affine_t, scalar_t> *ctx = context->context;
    assert(ctx->config.npoints == npoints);
    assert(ctx->ffi_affine_sz == ffi_affine_sz);
    assert(batches > 0);

    hipStream_t stream = ctx->pipp.default_stream;
    stream_t aux_stream(ctx->pipp.get_device());

    try
    {
        for (size_t i = 0; i < batches; i++)
        {
            out[i].inf();
        }

        typename pipp_t::result_container_t_faster *kernel_res = &ctx->fres0;
        typename pipp_t::result_container_t_faster *accum_res = &ctx->fres1;
        // 每次执行两批
        // 一批传送
        // 一批计算
        size_t d_scalars_xfer = ctx->d_scalars_sn[0];
        size_t d_scalars_compute = ctx->d_scalars_sn[1];

        channel_t<size_t> ch;
        size_t scalars_sz = ctx->pipp.get_size_scalars(ctx->config);

        int work = 0;
        // 复制第 0 批标量到h_scalars
        memcpy(ctx->h_scalars, &scalars[work * npoints], scalars_sz);
        // 把计算点传送到设备中
        ctx->pipp.transfer_scalars_to_device(ctx->config, d_scalars_compute,
                                             ctx->h_scalars, aux_stream);
        CUDA_OK(hipStreamSynchronize(aux_stream));

        for (; work < (int)batches; work++)
        {
            // Launch the GPU kernel, transfer the results back
            batch_pool.spawn([&]()
                             {

                CUDA_OK(hipStreamSynchronize(aux_stream));
                // 进行标量变换，{2^c}k_{i,j} => {2^{c-1}}k_{i,j} | sign 获得对应 point_idx
                // k1,1|sign k2,1|sign ...kn,1|sign , ... , k1,[lambda/c]|sign k2,[lambda/c]|sign ... kn,[lambda/c]|sign
                // p1,p2,.. pn    p1,p2,pn    p1,p2,pn ...
                //printf("begin launch_jy_process_scalar_1\n");
                ctx->pipp.launch_jy_process_scalar_1(ctx->config, d_scalars_compute,
                                                  ctx->jy_d_scalar_tuples_sn,
                                                  ctx->jy_d_point_idx_sn
                                                  );
                //printf("end launch_jy_process_scalar_1\n");

                // scalar point
                uint32_t* jy_d_scalar_tuple = ctx->pipp.jy_d_scalar_tuple_ptrs[ctx->jy_d_scalar_tuples_sn];
                uint32_t* jy_d_scalar_tuple_out = ctx->pipp.jy_d_scalar_tuple_ptrs[ctx->jy_d_scalar_tuples_out_sn];
                uint32_t* jy_d_point_idx = ctx->pipp.jy_d_point_idx_ptrs[ctx->jy_d_point_idx_sn];
                uint32_t* jy_d_point_idx_out = ctx->pipp.jy_d_point_idx_ptrs[ctx->jy_d_point_idx_out_sn];
                uint32_t nscalars = npoints;
                // 主要是为了获取空间大小
                void *d_temp = NULL;
                size_t temp_sort_size = 0;
                // 暂时先将最低1位到最高31位获取sij
                hipcub::DeviceRadixSort::SortPairs(d_temp, temp_sort_size,
                                                jy_d_scalar_tuple, jy_d_scalar_tuple_out,
                                                jy_d_point_idx, jy_d_point_idx_out, nscalars, 0, 31, stream);
                void *d_cub_sort = (void *)ctx->pipp.d_cub_ptrs[ctx->d_cub_sort_idx];
                //printf("begin hipcub::DeviceRadixSort::SortPairs\n");
                // 在每个窗口内进行排序
                for(size_t k = 0; k < NWINS; k++)
                {
                    size_t ptr = k * nscalars;
                    hipcub::DeviceRadixSort::SortPairs(d_cub_sort, temp_sort_size,
                                                    jy_d_scalar_tuple + ptr, jy_d_scalar_tuple_out + ptr,
                                                    jy_d_point_idx + ptr, jy_d_point_idx_out + ptr, nscalars, 0, 31, stream);
                }
                //printf("end hipcub::DeviceRadixSort::SortPairs\n");
                //printf("begin launch_jy_process_scalar_2\n");
                // 获得 bucket index
                ctx->pipp.launch_process_scalar_2(ctx->config,
                                                  ctx->jy_d_scalar_tuples_out_sn, ctx->d_bucket_idx_sn);

                //printf("end launch_jy_process_scalar_2\n");

                // accumulate parts of the buckets into static buffers.
                // 预计算点
                //printf("begin launch_bucket_acc\n");
                ctx->pipp.launch_bucket_acc(ctx->config, ctx->jy_d_scalar_tuples_out_sn,
                                            ctx->d_bucket_idx_sn, ctx->jy_d_point_idx_out_sn,
                                            ctx->d_pre_points_sn, ctx->d_buckets_sn,
                                            ctx->d_buckets_pre_sn, ctx->d_bucket_idx_pre_vector_sn,
                                            ctx->d_bucket_idx_pre_used_sn, ctx->d_bucket_idx_pre_offset_sn);
                //printf("end launch_bucket_acc\n");

                //printf("begin launch_bucket_agg_1\n");
                ctx->pipp.launch_bucket_agg_1(ctx->config, ctx->d_buckets_sn);
                //printf("end launch_bucket_agg_1\n");
                //printf("begin launch_bucket_agg_2\n");
                ctx->pipp.launch_bucket_agg_2(ctx->config, ctx->d_buckets_sn,ctx->d_res_sn,ctx->d_st_sn,ctx->d_sost_sn);
                //printf("end launch_bucket_agg_2\n");

                // ctx->pipp.synchronize_stream();
                //printf("begin transfer_res_to_host_faster\n");
                ctx->pipp.transfer_res_to_host_faster(*kernel_res, ctx->d_res_sn);
                //printf("end transfer_res_to_host_faster\n");
                ctx->pipp.synchronize_stream();
                
                ch.send(work); });

            // Transfer the next set of scalars, Faccumulate the previous result
            batch_pool.spawn([&]()
                             {
                // Start next scalar transfer
                if (work + 1 < (int)batches) {
                    // Copy into pinned memory
                    memcpy(ctx->h_scalars, &scalars[(work + 1) * npoints], scalars_sz);

                    ctx->pipp.transfer_scalars_to_device(ctx->config,
                                                         d_scalars_xfer, ctx->h_scalars,
                                                         aux_stream);
                }
                // Accumulate the previous result
                if (work - 1 >= 0) {
                    ctx->pipp.accumulate_faster(out[work - 1], *accum_res);

		    
                }
                ch.send(work); });
            ch.recv();
            ch.recv();
            std::swap(kernel_res, accum_res);
            std::swap(d_scalars_xfer, d_scalars_compute);
        }

        // Accumulate the final result
        ctx->pipp.accumulate_faster(out[batches - 1], *accum_res);
    }
    catch (const cuda_error &e)
    {
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
        return RustError{e.code(), e.what()};
#else
        return RustError { e.code() }
#endif
    }

    return RustError{hipSuccess};
}

#endif //  __CUDA_ARCH__
