#include "hip/hip_runtime.h"
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <sys/mman.h>
#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>
#include <ec/xyzt_t.hpp>
#include <util/log.h>
#include <ff/bls12-381.hpp>

// #if defined(FEATURE_BLS12_381)
// #include <ff/bls12-381.hpp>
// #elif defined(FEATURE_BLS12_377)
// #include <ff/bls12-377.hpp>
// #else
// #error "Unknown curve"
// #endif

typedef jacobian_t<fp_t> point_t;
typedef xyzz_t<fp_t> bucket_t;
// typedef xyzt_t<fp_t> bucket_t;
typedef bucket_t::affine_inf_t affine_t;
typedef fr_t scalar_t;

#include <msm/pippenger.cuh>

// init cub radix sort
extern "C" void cub_initial()
{
  uint32_t *d_scalar_tuple = nullptr;
  uint32_t *d_scalar_tuple_out = nullptr;
  uint32_t *d_point_idx = nullptr;
  uint32_t *d_point_idx_out = nullptr;
  uint32_t *d_offset_a = nullptr;
  uint32_t *d_offset_b = nullptr;
  void *d_temp = NULL;
  size_t temp_size = 0;
  hipcub::DeviceRadixSort::SortPairs(d_temp, temp_size, d_scalar_tuple,
                                  d_scalar_tuple_out, d_point_idx,
                                  d_point_idx_out, 1);
  hipcub::DeviceSegmentedRadixSort::SortPairs(
      d_temp, temp_size, d_scalar_tuple, d_scalar_tuple_out, d_point_idx,
      d_point_idx_out, 1, 1, d_offset_a, d_offset_b);
}

#ifndef __CUDA_ARCH__

// 每批次启动线程数
static const size_t NUM_BATCH_THREADS = 2;
static thread_pool_t batch_pool(NUM_BATCH_THREADS);

typedef pippenger_t<bucket_t, point_t, affine_t, scalar_t> pipp_t;

// MSM context used store persistent state
template <class bucket_t, class affine_t, class scalar_t>
struct Context
{
  pipp_t pipp;
  typename pipp_t::MSMConfig config;
  size_t ffi_affine_sz;
  size_t d_pre_points_sn;
  size_t d_scalars_sn[NUM_BATCH_THREADS];
  size_t d_buckets_sn;
  size_t d_scalar_tuples_sn;
  size_t d_point_idx_sn;
  size_t d_scalar_tuples_out_sn;
  size_t d_point_idx_out_sn;
  // buffer
  size_t d_buckets_pre_sn;
  // buffer index
  size_t d_bucket_idx_pre_vector_sn;
  // buffer used
  size_t d_bucket_idx_pre_used_sn;
  // buffer offest
  size_t d_bucket_idx_pre_offset_sn;
  size_t d_res_sn;
  size_t d_sost_sn;
  size_t d_cub_sort_idx;
  scalar_t *h_scalars;

  typename pipp_t::result_container_t_faster fres0;
  typename pipp_t::result_container_t_faster fres1;
};

template <class bucket_t, class affine_t, class scalar_t>
struct RustContext
{
  Context<bucket_t, affine_t, scalar_t> *context;
};

// Initialization function
// Allocate device storage, transfer bases
extern "C" RustError
mult_pippenger_faster_init(RustContext<bucket_t, affine_t, scalar_t> *context,
                           const affine_t points[], size_t npoints,
                           size_t ffi_affine_sz)
{
  LOG(INFO, "MSM init");
  context->context = new Context<bucket_t, affine_t, scalar_t>();
  Context<bucket_t, affine_t, scalar_t> *ctx = context->context;
  ctx->ffi_affine_sz = ffi_affine_sz;
  try
  {
    ctx->config = ctx->pipp.init_msm_faster(npoints);
    LOG(INFO, "Molloc MSM memory");
    ctx->d_pre_points_sn = ctx->pipp.allocate_d_pre_points(ctx->config);
    //
    for (size_t i = 0; i < NUM_BATCH_THREADS; i++)
    {
      ctx->d_scalars_sn[i] = ctx->pipp.allocate_d_scalars(ctx->config);
    }
    ctx->d_buckets_sn = ctx->pipp.allocate_d_buckets();
    ctx->d_buckets_pre_sn = ctx->pipp.allocate_d_buckets_pre(ctx->config);
    ctx->d_bucket_idx_pre_vector_sn =
        ctx->pipp.allocate_d_bucket_idx_pre_vector(ctx->config);
    ctx->d_bucket_idx_pre_used_sn =
        ctx->pipp.allocate_d_bucket_idx_pre_used(ctx->config);
    ctx->d_bucket_idx_pre_offset_sn =
        ctx->pipp.allocate_d_bucket_idx_pre_offset(ctx->config);

    ctx->d_sost_sn = ctx->pipp.allocate_d_sost(ctx->config);
    ctx->d_res_sn = ctx->pipp.allocate_d_res();
    ctx->d_scalar_tuples_sn =
        ctx->pipp.allocate_d_scalar_tuple(ctx->config);
    ctx->d_point_idx_sn = ctx->pipp.allocate_d_point_idx(ctx->config);
    ctx->d_scalar_tuples_out_sn =
        ctx->pipp.allocate_d_scalar_tuple_out(ctx->config);
    ctx->d_point_idx_out_sn = ctx->pipp.allocate_d_point_idx(ctx->config);
    ctx->d_cub_sort_idx = ctx->pipp.allocate_d_cub_sort_faster(ctx->config);

    // Allocate pinned memory on host
    CUDA_OK(hipHostMalloc(&ctx->h_scalars,
                           ctx->pipp.get_size_scalars(ctx->config)));

    LOG(INFO, "Transfer bases to device");

    ctx->pipp.transfer_bases_to_device(ctx->config, ctx->d_pre_points_sn,
                                       points, ffi_affine_sz);
    LOG(INFO, "Launch kernel pre compute init");
    ctx->pipp.launch_kernel_pre_compute_init(ctx->config, ctx->d_pre_points_sn);

    ctx->fres0 = ctx->pipp.get_result_container_faster();
    ctx->fres1 = ctx->pipp.get_result_container_faster();
  }
  catch (const cuda_error &e)
  {
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
    return RustError{e.code(), e.what()};
#else
    return RustError { e.code() }
#endif
  }
  return RustError{hipSuccess};
}

// Peform MSM on a batch of scalars over fixed bases
extern "C" RustError
mult_pippenger_faster_inf(RustContext<bucket_t, affine_t, scalar_t> *context,
                          point_t *out, const affine_t points[], size_t npoints,
                          size_t batches, const scalar_t scalars[],
                          size_t ffi_affine_sz)
{
  (void)points; // Silence unused param warning

  Context<bucket_t, affine_t, scalar_t> *ctx = context->context;
  assert(ctx->config.npoints == npoints);
  assert(ctx->ffi_affine_sz == ffi_affine_sz);
  assert(batches > 0);

  hipStream_t stream = ctx->pipp.default_stream;
  stream_t aux_stream(ctx->pipp.get_device());

  try
  {
    for (size_t i = 0; i < batches; i++)
    {
      out[i].inf();
    }

    typename pipp_t::result_container_t_faster *kernel_res = &ctx->fres0;
    typename pipp_t::result_container_t_faster *accum_res = &ctx->fres1;

    size_t d_scalars_xfer = ctx->d_scalars_sn[0];
    size_t d_scalars_compute = ctx->d_scalars_sn[1];

    channel_t<size_t> ch;
    size_t scalars_sz = ctx->pipp.get_size_scalars(ctx->config);

    int work = 0;
    LOG(INFO, "Transfer scalars to device");
    memcpy(ctx->h_scalars, &scalars[work * npoints], scalars_sz);
    ctx->pipp.transfer_scalars_to_device(ctx->config, d_scalars_compute,
                                         ctx->h_scalars, aux_stream);
    CUDA_OK(hipStreamSynchronize(aux_stream));

    for (; work < (int)batches; work++)
    {
      // Launch the GPU kernel, transfer the results back
      batch_pool.spawn([&]()
                       {
        CUDA_OK(hipStreamSynchronize(aux_stream));
        LOG(INFO, "Launch process scalars");
        nvtxRangePushA("process_scalars");
        ctx->pipp.launch_process_scalars(ctx->config, d_scalars_compute,
                                        ctx->d_scalar_tuples_sn,
                                        ctx->d_point_idx_sn);
        nvtxRangePop();
        // scalar point
        uint32_t *d_scalar_tuple =
            ctx->pipp.d_scalar_tuple_ptrs[ctx->d_scalar_tuples_sn];
        uint32_t *d_scalar_tuple_out =
            ctx->pipp.d_scalar_tuple_ptrs[ctx->d_scalar_tuples_out_sn];
        uint32_t *d_point_idx =
            ctx->pipp.d_point_idx_ptrs[ctx->d_point_idx_sn];
        uint32_t *d_point_idx_out =
            ctx->pipp.d_point_idx_ptrs[ctx->d_point_idx_out_sn];
        uint32_t nscalars = npoints;
        void *d_temp = NULL;
        size_t temp_sort_size = 0;
        // 暂时先将最低1位到最高31位获取sij
        hipcub::DeviceRadixSort::SortPairs(
            d_temp, temp_sort_size, d_scalar_tuple, d_scalar_tuple_out,
            d_point_idx, d_point_idx_out, nscalars, 0, 31, stream);
        void *d_cub_sort = (void *)ctx->pipp.d_cub_ptrs[ctx->d_cub_sort_idx];
        // 在每个窗口内进行排序
        LOG(INFO, "Launch sort");
        for (size_t k = 0; k < NWINS; k++) {
          size_t ptr = k * nscalars;
          hipcub::DeviceRadixSort::SortPairs(
              d_cub_sort, temp_sort_size, d_scalar_tuple + ptr,
              d_scalar_tuple_out + ptr, d_point_idx + ptr,
              d_point_idx_out + ptr, nscalars, 0, 31, stream);
        }

        // accumulate parts of the buckets into static buffers.
        LOG(INFO, "Launch bucket acc");
        nvtxRangePushA("bucket_acc");
        ctx->pipp.launch_bucket_acc(
            ctx->config, ctx->d_scalar_tuples_out_sn,
            ctx->d_point_idx_out_sn, ctx->d_pre_points_sn, ctx->d_buckets_sn,
            ctx->d_buckets_pre_sn, ctx->d_bucket_idx_pre_vector_sn,
            ctx->d_bucket_idx_pre_used_sn, ctx->d_bucket_idx_pre_offset_sn);
        nvtxRangePop();
        LOG(INFO, "Launch bucket agg");
        nvtxRangePushA("bucket_agg_1");

        ctx->pipp.launch_bucket_agg_1(ctx->config, ctx->d_buckets_sn);
        nvtxRangePop();
        nvtxRangePushA("bucket_agg_2");

        ctx->pipp.launch_bucket_agg_2(ctx->config, ctx->d_buckets_sn,
                                      ctx->d_res_sn, ctx->d_sost_sn);
        nvtxRangePop();
        LOG(INFO, "Transfer res to host");
        ctx->pipp.transfer_res_to_host_faster(*kernel_res, ctx->d_res_sn);
        ctx->pipp.synchronize_stream();

        ch.send(work); });

      // Transfer the next set of scalars, Faccumulate the previous result
      batch_pool.spawn([&]()
                       {
        // Start next scalar transfer
        if (work + 1 < (int)batches) {
          // Copy into pinned memory
          LOG(INFO, "Transfer next batch scalars to device");
          memcpy(ctx->h_scalars, &scalars[(work + 1) * npoints], scalars_sz);

          ctx->pipp.transfer_scalars_to_device(ctx->config, d_scalars_xfer,
                                               ctx->h_scalars, aux_stream);
        }
        // Accumulate the previous result
        if (work - 1 >= 0) {
          LOG(INFO, "Accumulate result");
          ctx->pipp.accumulate_faster(out[work - 1], *accum_res);
        }
        ch.send(work); });
      ch.recv();
      ch.recv();
      std::swap(kernel_res, accum_res);
      std::swap(d_scalars_xfer, d_scalars_compute);
    }

    // Accumulate the final result
    LOG(INFO, "Accumulate final result");
    ctx->pipp.accumulate_faster(out[batches - 1], *accum_res);
  }
  catch (const cuda_error &e)
  {
#ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
    return RustError{e.code(), e.what()};
#else
    return RustError { e.code() }
#endif
  }

  return RustError{hipSuccess};
}

#endif //  __CUDA_ARCH__
